#include <iostream>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <execution>

__device__ unsigned int numPrimesFound = 1;

__device__ inline bool estPremier(unsigned int const& n) {
    if (n % 2 == 0 || n % 3 == 0)
        return false;

    for (unsigned int i = 5; i * i <= n; i += 6)
    {
        if (n % i == 0)
            return false;

        if (n % (i + 2) == 0)
            return false;
    }
    return true;
}

__global__ void search_kernel(unsigned int *primes, unsigned int fin)
{
    int id = threadIdx.x;
    int stride = blockDim.x;

    if (id == 0) {
        primes[0] = 2;
    }
    __syncthreads();

    unsigned int n = 3 + id;
    printf("thread id :%d\n", id);
    printf("n = %d\n", n);
    while (true)
    {
        if (estPremier(n))
        {
            unsigned int index = atomicAdd((unsigned int*)&numPrimesFound, 1);
            if (index < fin)
                primes[index] = n;
            else
                break;
        }
        n += stride;

        // Synchronize threads and check if the desired number of primes is found
        __syncthreads();
        if (numPrimesFound >= fin) {
            break;
        }
    }
}

unsigned int* find_gpu(unsigned int const& fin)
{
    // Allocate memory on the device for primes
    unsigned int *d_primes;
    hipError_t err = hipMalloc(&d_primes, fin * sizeof(unsigned int));
    if (err != hipSuccess) {
        fprintf(stderr, "\nFailed to allocate device memory (error code %s)!\n", hipGetErrorString(err));
        return nullptr;
    }

    // Get the max number of blocks and threads
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int blockSize = prop.maxThreadsPerBlock;
    int numBlocks = (fin + blockSize - 1) / blockSize; // Calculate the number of blocks needed

    // Call the kernel
    search_kernel<<<1, 1>>>(d_primes, fin);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "\nFailed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        hipFree(d_primes);
        return nullptr;
    }

    // Synchronize the device
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "\nFailed to synchronize device (error code %s)!\n", hipGetErrorString(err));
        hipFree(d_primes);
        return nullptr;
    }

    // Copy the result back to the host
    unsigned int *primes = new unsigned int[fin];
    err = hipMemcpy(primes, d_primes, fin * sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy memory from device to host (error code %s)!\n", hipGetErrorString(err));
        hipFree(d_primes);
        delete[] primes;
        return nullptr;
    }

    std::sort(primes, primes + fin);

    // Free the memory on the device
    hipFree(d_primes);

    return primes;
}
