#include <iostream>
#include <hip/hip_runtime.h>

__device__ volatile unsigned int numPrimesFound = 1;

__device__ inline bool estPremier(unsigned int const& n) {
    for (unsigned int i = 2; i * i <= n; i++)
        if (n % i == 0)
            return false;
    
    return true;
}

__global__ void search_kernel(unsigned int *primes, unsigned int fin)
{
    int id = threadIdx.x;
    int stride = blockDim.x;

    if (id == 0) {
        primes[0] = 2;
    }
    __syncthreads();

    int n = 3 + id;
    while (true)
    {
        if (estPremier(n))
        {
            unsigned int index = atomicAdd((unsigned int*)&numPrimesFound, 1);
            primes[index] = n;
        }
        n += stride;

        // Synchronize threads and check if the desired number of primes is found
        if (numPrimesFound >= fin) {
            break;
        }
    }
}

unsigned int* find_gpu(unsigned int const& fin)
{
    // Allocate memory on the device for primes
    unsigned int *d_primes;
    hipError_t err = hipMalloc(&d_primes, fin * sizeof(unsigned int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory (error code %s)!\n", hipGetErrorString(err));
        return nullptr;
    }
    
    // Get the max number of blocks and threads
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int blockSize = prop.maxThreadsPerBlock;
    int numBlocks = (fin + blockSize - 1) / blockSize; // Calculate the number of blocks needed
    
    // Call the kernel
    search_kernel<<<numBlocks, blockSize>>>(d_primes, fin);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        hipFree(d_primes);
        return nullptr;
    }
    
    // Synchronize the device
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to synchronize device (error code %s)!\n", hipGetErrorString(err));
        hipFree(d_primes);
        return nullptr;
    }
    
    // Copy the result back to the host
    unsigned int *primes = new unsigned int[fin];
    err = hipMemcpy(primes, d_primes, fin * sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy memory from device to host (error code %s)!\n", hipGetErrorString(err));
        hipFree(d_primes);
        delete[] primes;
        return nullptr;
    }
    
    // Free the memory on the device
    hipFree(d_primes);
    
    return primes;
}