
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <cstring>
#include <stdexcept>

typedef unsigned int uint;

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << ": " << hipGetErrorString(err) << std::endl;
        // In a real application, you might throw an exception or exit.
        exit(EXIT_FAILURE);
    }
}

__global__ void find_kernel(uint fin, bool *isPrimes)
{
    // Each block tests a different number 'i' in a grid-striding loop.
    for (uint i = 3 + blockIdx.x; i <= fin; i += gridDim.x)
    {
        // 1. Correctly declare and initialize a shared flag for the block.
        //    Only one thread needs to do the initialization.
        __shared__ bool isPrime_flag;
        if (threadIdx.x == 0) {
            isPrime_flag = true;
        }
        // Synchronize to make sure isPrime_flag is visible to all threads.
        //__syncthreads();

        // 2. Perform parallel trial division.
        //    Let all threads check their assigned divisors.
        //    The loop condition is now much simpler and safer.
        //    The check is j*j <= i, not i/2
        for (uint j = 3 + threadIdx.x; j * j <= i; j += blockDim.x)
        {
            // If the flag is already false, we can skip the check. This is a safe optimization.
            if (!isPrime_flag) {
                break;
            }

            if (i % j == 0) {
                // A divisor is found. Set the shared flag to false.
                // This is a benign race condition, as all threads write the same value (false).
                isPrime_flag = false;
            }
        }

        // 3. Synchronize to ensure the final result of isPrime_flag is visible to all threads.
        //__syncthreads();

        // 4. Have ONLY ONE thread write the result to global memory to avoid a race condition.
        if (threadIdx.x == 0 && isPrime_flag == true)
        {
            // Assuming isPrimes maps odd numbers. If i=3, idx=0. If i=5, idx=1. So (i-3)/2
            isPrimes[(i - 3) / 2] = true;
        }
    }
}

uint *find_to_n(uint const& fin, uint &numPrimesFound) {
    // Handle cases where 0 primes are requested.
    if (fin < 2) return NULL;

    const uint ARRAY_SIZE = (fin / 2) - 1;

    // initialisation du tableau
    bool *isPrimes = new bool[ARRAY_SIZE];
    memset(isPrimes, 0, (ARRAY_SIZE) * sizeof(bool));
    
    bool * isPrimes_d;
    // Allocate memory on the device for the boolean array
    checkCudaError(hipMalloc(&isPrimes_d, ARRAY_SIZE * sizeof(bool)), "hipMalloc isPrimes_d failed");
    // Copy initial (all false) data from host to device
    checkCudaError(hipMemcpy(isPrimes_d, isPrimes, ARRAY_SIZE * sizeof(bool), hipMemcpyHostToDevice), "hipMemcpy isPrimes H2D failed");

    // Get device properties for kernel launch configuration
    int deviceId;
    checkCudaError(hipGetDevice(&deviceId), "hipGetDevice failed"); // Get the current device ID

    hipDeviceProp_t deviceProp;
    checkCudaError(hipGetDeviceProperties(&deviceProp, deviceId), "hipGetDeviceProperties failed");

    // 1. Choose threads per block.
    int threadsPerBlock = deviceProp.maxThreadsPerBlock;

    // 2. Calculate blocks per grid.
    int blocksPerGrid = (ARRAY_SIZE + threadsPerBlock - 1) / threadsPerBlock;

    // Optional: Check if blocksPerGrid exceeds the maxGridSize for the x-dimension
    if (blocksPerGrid > deviceProp.maxGridSize[0]) {
        std::cerr << "Warning: Calculated blocksPerGrid (" << blocksPerGrid
                  << ") exceeds device's maxGridSize[0] (" << deviceProp.maxGridSize[0]
                  << "). Capping grid size." << std::endl;
        blocksPerGrid = deviceProp.maxGridSize[0]; // Cap it if it exceeds
    }

    // Launch the kernel!
    
    find_kernel<<<blocksPerGrid, threadsPerBlock, sizeof(bool)>>>(fin, isPrimes_d);
    checkCudaError(hipGetLastError(), "find_kernel launch failed"); // Check for errors immediately after launch

    // Synchronize to ensure the kernel finishes execution
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize failed");

    // Copy results back from device to host
    checkCudaError(hipMemcpy(isPrimes, isPrimes_d, ARRAY_SIZE * sizeof(bool), hipMemcpyDeviceToHost), "hipMemcpy D2H failed");

    // transform to int array
    uint *primes = new uint[fin];
    primes[0] = 2;
    numPrimesFound = 1;
    for (int i = 0; i < ARRAY_SIZE; i++) {
        if (!isPrimes[i]) continue;
        primes[numPrimesFound] = i * 2 + 3;
        numPrimesFound++;
    }

    delete [] isPrimes;
    hipFree(isPrimes_d);
    return primes;
}

inline bool estPremier(uint const& n, uint *primes) {
    for (uint i = 2; primes[i] * primes[i] <= n; i++)
        if (n % primes[i] == 0)
            return false;
    
    return true;
}

uint *find_n_primes(uint const& fin) {
    // Handle cases where 0 primes are requested.
    if (fin == 0) return NULL;

    // variables
    uint *primes;
    uint numPrimesFound = 2;

    // allocation dynamique de mémoire
    if ((primes = (uint*)malloc((fin + 1) * sizeof(uint))) == NULL)
        return NULL;
    primes[0] = 2;
    if (fin == 1) return primes;

    primes[1] = 3;
    if (fin == 2) return primes;

    primes[2] = 5;
    uint nTest = 5;
    while (numPrimesFound < fin){
        if (estPremier(nTest, primes))
        {
            primes[numPrimesFound] = nTest;
            numPrimesFound++;
        }
        if (estPremier(nTest + 2, primes))
        {
            primes[numPrimesFound] = nTest + 2;
            numPrimesFound++;
        }
        
        nTest += 6;
    }
    
    return primes;
}