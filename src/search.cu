#include <iostream>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <execution>

__device__ unsigned int numPrimesFound = 2;

__device__ inline bool estPremier(unsigned int const& n) {
    if (n % 3 == 0) return false;

    // Check for divisibility by numbers of the form 6k ± 1 up to sqrt(n)
    
    for (unsigned int i = 5; i * i <= n; i += 6)
    {
        if (n % i == 0)
            return false;

        if (n % (i + 2) == 0)
            return false;
    }
    return true;
}

__global__ void search_kernel(unsigned int *primes, unsigned int fin)
{
    int id = threadIdx.x;
    int stride = blockDim.x * 2;

    unsigned int n = 5 + id * 2;
    while (numPrimesFound < fin)
    {
        if (estPremier(n))
        {
            unsigned int index = atomicAdd((unsigned int*)&numPrimesFound, 1);
            primes[index] = n;
        }
        n += stride;
        __syncthreads();
    }
}

unsigned int* find(unsigned int const& fin)
{
    // Allocate memory on the device for primes
    unsigned int *d_primes;
    hipError_t err = hipMalloc(&d_primes, fin * sizeof(unsigned int));
    if (err != hipSuccess) {
        fprintf(stderr, "\nFailed to allocate device memory (error : %s)!\n", hipGetErrorString(err));
        return nullptr;
    }

    unsigned int initials[3] = { 2, 3, 5 };

    // Copy initial primes to the device
    err = hipMemcpy(d_primes, &initials[0], sizeof(initials), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "\nFailed to copy initial primes to the device memory (error : %s)!\n", hipGetErrorString(err));
        return nullptr;
    }

    // Get the max number of blocks and threads
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int blockSize = prop.maxThreadsPerBlock;
    int numBlocks = (fin + blockSize) / blockSize; // Calculate the number of blocks needed

    if (blockSize > fin)
        blockSize = fin;

    // Call the kernel
    search_kernel<<<numBlocks, blockSize>>>(d_primes, fin);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "\nFailed to launch kernel (error : %s)!\n", hipGetErrorString(err));
        hipFree(d_primes);
        return nullptr;
    }

    // Synchronize the device
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "\nFailed to synchronize device (error : %s)!\n", hipGetErrorString(err));
        hipFree(d_primes);
        return nullptr;
    }

    // Copy the result back to the host
    unsigned int *primes = new unsigned int[fin];
    err = hipMemcpy(primes, d_primes, fin * sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy memory from device to host (error : %s)!\n", hipGetErrorString(err));
        hipFree(d_primes);
        delete[] primes;
        return nullptr;
    }

    std::sort(primes, primes + fin);

    // Free the memory on the device
    hipFree(d_primes);

    return primes;
}
