
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <cstring>
#include <stdexcept>

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << ": " << hipGetErrorString(err) << std::endl;
        // In a real application, you might throw an exception or exit.
        exit(EXIT_FAILURE);
    }
}

__device__ unsigned int nbPrimesFound = 1;

__device__ bool estPremier(unsigned int const& n)
{
    if (n <= 1) return false;
    if (n <= 3) return true;

    if (n % 2 == 0) return false;
    if (n % 3 == 0) return false;

    for (unsigned int i = 5; i * i < n; i += 6)
    {
        if (n % i == 0 || n % (i + 2) == 0)
            return false;
    }
    return true;
}

__global__ void find_kernel(unsigned int fin, bool *isPrimes)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;;
    unsigned int nbPrimesFound_local = 0;

    for (unsigned int i = idx; i < fin / 2; i += gridDim.x * blockDim.x){
        if (estPremier(i * 2 + 3))
        {
            isPrimes[i] = true;
            nbPrimesFound_local++;
        }
    }

    atomicAdd(&nbPrimesFound, nbPrimesFound_local);
}

unsigned int *find_to_n(unsigned int const& fin, unsigned int &numPrimesFound) {
    // Handle cases where 0 primes are requested.
    if (fin < 2) return NULL;

    const unsigned int ARRAY_SIZE = (fin / 2) - 1;

    // initialisation du tableau
    bool *isPrimes = new bool[ARRAY_SIZE];
    memset(isPrimes, 0, (ARRAY_SIZE) * sizeof(bool));
    
    bool * isPrimes_d;
    // Allocate memory on the device for the boolean array
    checkCudaError(hipMalloc(&isPrimes_d, ARRAY_SIZE * sizeof(bool)), "hipMalloc isPrimes_d failed");
    // Copy initial (all false) data from host to device
    checkCudaError(hipMemcpy(isPrimes_d, isPrimes, ARRAY_SIZE * sizeof(bool), hipMemcpyHostToDevice), "hipMemcpy isPrimes H2D failed");

    // Get device properties for kernel launch configuration
    int deviceId;
    checkCudaError(hipGetDevice(&deviceId), "hipGetDevice failed"); // Get the current device ID

    hipDeviceProp_t deviceProp;
    checkCudaError(hipGetDeviceProperties(&deviceProp, deviceId), "hipGetDeviceProperties failed");

    // 1. Choose threads per block.
    int threadsPerBlock = 1024;
    if (threadsPerBlock > deviceProp.maxThreadsPerBlock) {
        threadsPerBlock = deviceProp.maxThreadsPerBlock;
        std::cerr << "Warning: threadsPerBlock adjusted to device max: " << threadsPerBlock << std::endl;
    }

    // 2. Calculate blocks per grid.
    int blocksPerGrid = (ARRAY_SIZE + threadsPerBlock - 1) / threadsPerBlock;

    // Optional: Check if blocksPerGrid exceeds the maxGridSize for the x-dimension
    if (blocksPerGrid > deviceProp.maxGridSize[0]) {
        std::cerr << "Warning: Calculated blocksPerGrid (" << blocksPerGrid
                  << ") exceeds device's maxGridSize[0] (" << deviceProp.maxGridSize[0]
                  << "). Capping grid size." << std::endl;
        blocksPerGrid = deviceProp.maxGridSize[0]; // Cap it if it exceeds
    }

    // Launch the kernel!
    find_kernel<<<blocksPerGrid, threadsPerBlock>>>(fin, isPrimes_d);
    checkCudaError(hipGetLastError(), "find_kernel launch failed"); // Check for errors immediately after launch

    // Synchronize to ensure the kernel finishes execution
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize failed");

    // Copy results back from device to host
    checkCudaError(hipMemcpy(isPrimes, isPrimes_d, ARRAY_SIZE * sizeof(bool), hipMemcpyDeviceToHost), "hipMemcpy D2H failed");
    
    // Copy the final prime count from device global memory to host
    checkCudaError(hipMemcpyFromSymbol(&numPrimesFound, HIP_SYMBOL(nbPrimesFound), sizeof(unsigned int)), "hipMemcpyFromSymbol nbPrimesFound failed");

    // transform to int array
    unsigned int *primes = new unsigned int[fin];
    primes[0] = 2;
    numPrimesFound = 1;
    for (int i = 0; i < ARRAY_SIZE; i++) {
        if (!isPrimes[i]) continue;
        primes[numPrimesFound] = i * 2 + 3;
        numPrimesFound++;
    }

    delete [] isPrimes;
    hipFree(isPrimes_d);
    return primes;
}

inline bool estPremier(unsigned int const& n, unsigned int *primes) {
    for (unsigned int i = 2; primes[i] * primes[i] <= n; i++)
        if (n % primes[i] == 0)
            return false;
    
    return true;
}

unsigned int *find_n_primes(unsigned int const& fin) {
    // Handle cases where 0 primes are requested.
    if (fin == 0) return NULL;

    // variables
    unsigned int *primes;
    unsigned int numPrimesFound = 2;

    // allocation dynamique de mémoire
    if ((primes = (unsigned int*)malloc((fin + 1) * sizeof(unsigned int))) == NULL)
        return NULL;
    primes[0] = 2;
    if (fin == 1) return primes;

    primes[1] = 3;
    if (fin == 2) return primes;

    primes[2] = 5;
    unsigned int nTest = 5;
    while (numPrimesFound < fin){
        if (estPremier(nTest, primes))
        {
            primes[numPrimesFound] = nTest;
            numPrimesFound++;
        }
        if (estPremier(nTest + 2, primes))
        {
            primes[numPrimesFound] = nTest + 2;
            numPrimesFound++;
        }
        
        nTest += 6;
    }
    
    return primes;
}