#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <time.h>

#ifdef _WIN32
#include <windows.h>
#endif

#include <hip/hip_runtime.h>


#define MAX_SIZE 2000000000

long *liste, *liste_d;
int nombresPremiers, *nombresPremiers_d;

__global__ void thread(long fin, int *nombresPremiersTrouves, long *liste){
    int tid = blockIdx.x * blockDim.x + threadIdx.x + 2;
    int estPremier;

    while (true) {
        estPremier = 1;
        for (long j = 2; j * j <= tid; j++) {
            if (tid % j == 0){
                estPremier = 0;
                break;
            }
        }
        if (estPremier) {
            int idx = atomicAdd(nombresPremiersTrouves, 1);
            if (idx < fin) {
                liste[idx] = tid;
            }
        }
        tid += gridDim.x * blockDim.x;

        if (*nombresPremiersTrouves >= fin) {
            break;
        }
    }
}

int cmpfunc (const void * a, const void * b) {
   return ( *(int*)a - *(int*)b );
}

int main() {
    #ifdef _WIN32
    SetConsoleOutputCP(CP_UTF8);
    #endif
    //demander à l'utilisateur combien de nombres veut-il chercher et enregistrer la réponse dans fin
    printf("Combien de nombres premiers voulez-vous chercher ? (max: 2 000 000 000)");
    long fin, *fin_d;
    scanf("%ld", &fin);
    if (fin <= MAX_SIZE) {
        printf("\033[2J\033[H");
        hipMalloc(&liste_d, fin * sizeof(long));
        hipMalloc(&fin_d, sizeof(long));
        hipMalloc(&nombresPremiers_d, sizeof(long));
        if (liste_d != NULL)
        {
            time_t startTime, stop;
	        startTime = time(NULL);

            int *nombresPremiers_d;
            hipMalloc(&nombresPremiers_d, sizeof(int));

            int init = 0;
            hipMemcpy(nombresPremiers_d, &init, sizeof(int), hipMemcpyHostToDevice);

            int block_size = 256;
            int grid_size = ((fin + block_size) / block_size);
            thread<<<grid_size,block_size>>>(fin, nombresPremiers_d, liste_d);

            liste = (long*)malloc(fin * sizeof(long));

            if (liste == NULL)
            {
                printf("\033[0;31m");
                printf("Impossible d'effectuer l'allocation dynamique de mémoire sur le CPU\n");
                printf("\033[0;37m");
                exit(EXIT_FAILURE);
            }

	        hipMemcpy(liste, liste_d, fin * sizeof(long), hipMemcpyDeviceToHost);

            hipFree(nombresPremiers_d);
            hipFree(liste_d);

            stop = time(NULL);
            printf("\033[2J\033[1;1H");
            printf("La recherche est terminée. En %lld secondes.\n\t1. Enregistrer dans Nombres-Premiers.txt.\n\t2. Tout afficher\n\t3. Afficher et Enregistrer\n:", stop - startTime);
            int rep;
            scanf("%d", &rep);
            if (rep == 1)
            {
                qsort(liste, fin, sizeof(long), cmpfunc);
                FILE *fichier = fopen("Nombres-Premiers.txt", "w+");
                if (fichier != NULL)
                {
                    for (long i = 0; i < fin; i++)
                    {
                        fprintf(fichier, "%ld\n", liste[i]);
                    }
                }
            }
            else if (rep == 2)
            {
                qsort(liste, fin, sizeof(long), cmpfunc);
                for (long i = 0; i < fin; i++)
                {
                    printf("%ld\n", liste[i]);
                }
            }
            else if (rep == 3)
            {
                qsort(liste, fin, sizeof(long), cmpfunc);
                FILE *fichier = fopen("Nombres-Premiers.txt", "w+");
                if (fichier != NULL)
                {
                    for (long i = 0; i < fin; i++)
                    {
                        fprintf(fichier, "%ld\n", liste[i]);
                    }
                }
                for (long i = 0; i < fin; i++)
                {
                    printf("%ld\n", liste[i]);
                }
            }
            else
            {
                printf("Veuillez entrer une réponse correct(1/2/3).\n");
                exit(EXIT_FAILURE);
            }
            free(liste);
        }
        else{
            printf("\033[0;31m");
            printf("Erreur! Impossible d'effectuer l'allocation dynamique de mémoire sur le GPU\n");
	        printf("\033[0;37m");
            exit(EXIT_FAILURE);
        }
    } else {
	    printf("\033[0;31m");
        printf("Erreur! Votre nombre est incorrect, veuillez entrer un nombre plus petit que %d\n", MAX_SIZE);
	    printf("\033[0;37m");
        exit(EXIT_FAILURE);
    }

    return 0;
}