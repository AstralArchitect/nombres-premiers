#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <time.h>

#ifdef _WIN32
#include <windows.h>
#endif

#include <hip/hip_runtime.h>


#define MAX_SIZE 2000000000

unsigned long long *liste, *liste_d;
int nombresPremiers, *nombresPremiers_d;

#ifdef _WIN32
void sleep_ms(DWORD milliseconds) {
    Sleep(milliseconds);
}
#else
void sleep_ms(unsigned long long milliseconds) {
    struct timespec ts;

    ts.tv_sec = milliseconds / 1000;
    ts.tv_nsec = (milliseconds % 1000) * 1000000;

    nanosleep(&ts, NULL);
}
#endif

__global__ void thread(unsigned long long fin, int *nombresPremiersTrouves, unsigned long long *liste){
    int tid = blockIdx.x * blockDim.x + threadIdx.x + 2;
    int estPremier;

    while (true) {
        estPremier = 1;
        for (unsigned long long j = 2; j * j <= tid; j++) {
            if (tid % j == 0){
                estPremier = 0;
                break;
            }
        }
        if (estPremier) {
            int idx = atomicAdd(nombresPremiersTrouves, 1);
            if (idx < fin) {
                liste[idx] = tid;
            }
        }
        tid += gridDim.x * blockDim.x;

        if (*nombresPremiersTrouves >= fin) {
            break;
        }
    }
}

int cmpfunc (const void * a, const void * b) {
   return ( *(int*)a - *(int*)b );
}

int main() {
    #ifdef _WIN32
    SetConsoleOutputCP(CP_UTF8);
    #endif
    //demander à l'utilisateur combien de nombres veut-il chercher et enregistrer la réponse dans fin
    printf("Combien de nombres premiers voulez-vous chercher ? (max: 2 000 000 000)");
    unsigned long long fin, *fin_d;
    scanf("%ld", &fin);
    if (fin <= MAX_SIZE) {
        printf("\033[2J\033[H");
        if (fin > 500000000000)
        {
            printf("Attention, la recherche risque de durer un certain temps\n");
            sleep_ms(2000);
        }

        printf("Recherche...\n");
        
        hipMalloc(&liste_d, fin * sizeof(unsigned long long));
        hipMalloc(&fin_d, sizeof(unsigned long long));
        hipMalloc(&nombresPremiers_d, sizeof(unsigned long long));
        if (liste_d != NULL)
        {
            time_t startTime, stop, searchStop;
	        startTime = time(NULL);

            int *nombresPremiers_d;
            hipMalloc(&nombresPremiers_d, sizeof(int));

            int init = 0;
            hipMemcpy(nombresPremiers_d, &init, sizeof(int), hipMemcpyHostToDevice);

            int block_size = 256;
            int grid_size = ((fin + block_size) / block_size);
            thread<<<grid_size,block_size>>>(fin, nombresPremiers_d, liste_d);

            liste = (unsigned long long*)malloc(fin * sizeof(unsigned long long));

            if (liste == NULL)
            {
                printf("\033[0;31m");
                printf("Impossible d'effectuer l'allocation dynamique de mémoire sur le CPU\n");
                printf("\033[0;37m");
                exit(EXIT_FAILURE);
            }

	        hipMemcpy(liste, liste_d, fin * sizeof(unsigned long long), hipMemcpyDeviceToHost);

            searchStop = time(NULL);

            hipFree(nombresPremiers_d);
            hipFree(liste_d);

            printf("\033[2J\033[1;1H");
            printf("Triage de la liste...\n");
            qsort(liste, fin, sizeof(int), cmpfunc);
            stop = time(NULL);
            printf("\033[2J\033[1;1H");
            printf("La recherche est terminée en %lld secondes. Le triage en %lld seconde. Total : %lld secondes\n\t1. Enregistrer dans Nombres-Premiers.txt.\n\t2. Tout afficher\n\t3. Afficher et Enregistrer\n:", searchStop - startTime, (stop - startTime) - (searchStop - startTime), stop - startTime);
            int rep;
            scanf("%d", &rep);
            if (rep == 1 || rep == 3)
            {
                FILE *fichier = fopen("Nombres-Premiers.txt", "w+");
                if (fichier != NULL)
                {
                    for (int i = 0; i < fin; i++)
                    {
                        fprintf(fichier, "%ld, ", liste[i]);
                    }
                }
            }
            else if (rep == 2 || rep == 3)
            {
                for (int i = 0; i < fin; i++)
                {
                    printf("%ld\n", liste[i]);
                }
            }
            else
            {
                printf("Veuillez entrer une réponse correct(1/2/3).\n");
                exit(EXIT_FAILURE);
            }
            free(liste);
        }
        else{
            printf("\033[0;31m");
            printf("Erreur! Impossible d'effectuer l'allocation dynamique de mémoire sur le GPU\n");
	        printf("\033[0;37m");
            exit(EXIT_FAILURE);
        }
    } else {
	    printf("\033[0;31m");
        printf("Erreur! Votre nombre est incorrect, veuillez entrer un nombre plus petit que %d\n", MAX_SIZE);
	    printf("\033[0;37m");
        exit(EXIT_FAILURE);
    }

    return 0;
}