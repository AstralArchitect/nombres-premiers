#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <time.h>

#ifdef _WIN32
#include <windows.h>
#endif

#include <hip/hip_runtime.h>


__device__ bool estPremier(unsigned long n)
{
    for (unsigned long j = 2; j * j <= n; j++) {
        if (n % j == 0){
            return false;
            break;
        }
    }
    return true;
}

__global__ void thread(unsigned long fin, unsigned long *nombresPremiersTrouves, unsigned long *liste){
    int tid = blockIdx.x * blockDim.x + threadIdx.x + 2;

    while (true) {
        if (estPremier(tid)) {
            int idx = atomicAdd((int*)nombresPremiersTrouves, 1);
            if (idx < fin) {
                liste[idx] = tid;
            }
        }
        tid += gridDim.x * blockDim.x;

        if (*nombresPremiersTrouves >= fin) {
            break;
        }
    }
}

unsigned long *find(unsigned long fin) {
    // host variables
    unsigned long *liste, nombresPremiers = 0;
    //device variables
    unsigned long *liste_d, *nombresPremiers_d;

    // allocation dynamique de mémoire sur le GPU
    hipMalloc(&liste_d, fin * sizeof(unsigned long));
    if (liste_d == NULL)
    {
        return NULL;
    }
    hipMalloc(&nombresPremiers_d, sizeof(unsigned long));
    if (nombresPremiers_d == NULL)
    {
        hipFree(liste_d);
        return NULL;
    }

    //copie de mémoire sur le GPU
    hipMemcpy(nombresPremiers_d, &nombresPremiers, sizeof(unsigned long), hipMemcpyHostToDevice);

    //apele des threads GPU
    int block_size = 512;
    int grid_size = ((fin + block_size) / block_size);
    thread<<<grid_size,block_size>>>(fin, nombresPremiers_d, liste_d);

    // allocation dynamique de mémoire sur le host
    liste = (unsigned long*)malloc(fin * sizeof(unsigned long));

    if (liste == NULL)
    {
        hipFree(nombresPremiers_d);
        hipFree(liste_d);
        return NULL;
    }

    // copie du resultat de la recherche sur le host
	hipMemcpy(liste, liste_d, fin * sizeof(unsigned long), hipMemcpyDeviceToHost);

    // liberation de la memoire allouée sur le GPU
    hipFree(nombresPremiers_d);
    hipFree(liste_d);

    return liste;
}