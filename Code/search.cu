#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>

#ifdef _WIN32
#include <windows.h>
#endif

#include <hip/hip_runtime.h>


__device__ bool cuda_estPremier(unsigned long n)
{
    if (n <= 3) {
        return true;
    }
    if (n % 2 == 0 || n % 3 == 0) {
        return false;
    }
    int i = 5;
    while (i * i <= n) {
        if (n % i == 0 || n % (i + 2) == 0) {
            return false;
        }
        i += 6;
    }
    return true;
}

__global__ void thread(unsigned long fin, unsigned long *nombresPremiersTrouves, unsigned long *liste){
    int tid = blockIdx.x * blockDim.x + threadIdx.x + 2;

    while (true) {
        if (cuda_estPremier(tid)) {
            int idx = atomicAdd((int*)nombresPremiersTrouves, 1);
            if (idx < fin) {
                liste[idx] = tid;
            }
        }
        tid += gridDim.x * blockDim.x;

        if (*nombresPremiersTrouves >= fin) {
            break;
        }
    }
}

extern "C" unsigned long *find(unsigned long fin) {
    // host variables
    unsigned long *liste, nombresPremiers = 0;
    //device variables
    unsigned long *liste_d, *nombresPremiers_d;

    // allocation dynamique de mémoire sur le GPU
    hipMalloc(&liste_d, fin * sizeof(unsigned long));
    hipMalloc(&nombresPremiers_d, sizeof(unsigned long));
    if (nombresPremiers_d == NULL)
    {
        hipFree(liste_d);
        return NULL;
    }

    //copie de mémoire sur le GPU
    hipMemcpy(nombresPremiers_d, &nombresPremiers, sizeof(unsigned long), hipMemcpyHostToDevice);

    //apele des threads GPU
    int block_size = 1024;
    int grid_size = ((fin + block_size) / block_size);
    thread<<<grid_size,block_size>>>(fin, nombresPremiers_d, liste_d);

    // allocation dynamique de mémoire sur le host
    liste = (unsigned long*)malloc(fin * sizeof(unsigned long));

    if (liste == NULL)
    {
        hipFree(nombresPremiers_d);
        hipFree(liste_d);
        return NULL;
    }

    // copie du resultat de la recherche sur le host
	hipMemcpy(liste, liste_d, fin * sizeof(unsigned long), hipMemcpyDeviceToHost);

    // liberation de la memoire allouée sur le GPU
    hipFree(nombresPremiers_d);
    hipFree(liste_d);

    return liste;
}