#include <stdio.h>
#include <stdbool.h>
#include <stdlib.h>
#include <time.h>

#ifdef _WIN32
#include <windows.h>
#endif

#include <hip/hip_runtime.h>


//device variables
unsigned long *liste_d, *nombresPremiers_d;

__device__ bool estPremier(unsigned long n)
{
    for (unsigned long j = 2; j * j <= n; j++) {
        if (n % j == 0){
            return false;
            break;
        }
    }
    return true;
}

__global__ void thread(unsigned long fin, unsigned long *nombresPremiersTrouves, unsigned long *liste){
    int tid = blockIdx.x * blockDim.x + threadIdx.x + 2;

    while (true) {
        if (estPremier(tid)) {
            int idx = atomicAdd((int*)nombresPremiersTrouves, 1);
            if (idx < fin) {
                liste[idx] = tid;
            }
        }
        tid += gridDim.x * blockDim.x;

        if (*nombresPremiersTrouves >= fin) {
            break;
        }
    }
}

unsigned long *find(unsigned long fin, unsigned long *nombresPremiers) {
    // host variables
    unsigned long *liste;

    // allocation dynamique de mémoire sur le GPU
    hipMalloc(&liste_d, fin * sizeof(unsigned long));
    hipMalloc(&nombresPremiers_d, sizeof(unsigned long));
    if (liste_d == NULL || nombresPremiers_d == NULL)
    {
        return NULL;
    }

    //copie de mémoire sur le GPU
    hipMemcpy(nombresPremiers_d, nombresPremiers, sizeof(unsigned long), hipMemcpyHostToDevice);

    //apele des threads GPU
    int block_size = 1024;
    int grid_size = ((fin + block_size) / block_size);
    thread<<<grid_size,block_size>>>(fin, nombresPremiers_d, liste_d);

    // allocation dynamique de mémoire sur le host
    liste = (unsigned long*)malloc(fin * sizeof(unsigned long));

    if (liste == NULL)
    {
        return NULL;
    }

    // copie du resultat de la recherche sur le host
	hipMemcpy(liste, liste_d, fin * sizeof(unsigned long), hipMemcpyDeviceToHost);

    // liberation de la memoire allouée sur le GPU
    hipFree(nombresPremiers_d);
    hipFree(liste_d);

    return liste;
}