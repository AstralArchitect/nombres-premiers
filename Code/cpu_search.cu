
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>

bool cpu_estPremier(unsigned long n)
{
    for (unsigned long j = 2; j * j <= n; j++) {
        if (n % j == 0){
            return false;
            break;
        }
    }
    return true;
}

unsigned long *find_cpu(unsigned long fin)
{
    unsigned long *liste = (unsigned long*)malloc(fin * sizeof(unsigned long));
    if (liste == NULL)
    {
        return NULL;
    }

    liste[0] = 2;

    unsigned long nombresPremiersTrouves = 0;
    for (int i = 3; nombresPremiersTrouves <= fin; i++) {
        if (cpu_estPremier(i)) {
            nombresPremiersTrouves++;
            liste[nombresPremiersTrouves] = i;
        }
    }

    return liste;
}